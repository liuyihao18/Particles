#include "hip/hip_runtime.h"
/*
 * reference: CUDA samples (particles)
 */

#include "kernel.cuh"

#include <hip/hip_runtime.h>   
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

extern "C" {

    void cudaInit(int argc, char** argv) {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest
        // Gflops/s
        devID = findCudaDevice(argc, (const char**)argv);

        if (devID < 0) {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void allocateArray(void** devPtr, uint size) {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void initializeArray(void* devPtr, uint size) {
        checkCudaErrors(hipMemset(devPtr, 0, size));
    }

    void freeArray(void* devPtr) {
        checkCudaErrors(hipFree(devPtr));
    }

    void copyArrayFromDevice(void* host, const void* device, uint size) {
        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    }

    void copyArrayToDevice(void* device, const void* host, int offset, int size) {
        checkCudaErrors(
            hipMemcpy((char*)device + offset, host, size, hipMemcpyHostToDevice));
    }

    void setParameters(SimParams* hostParams) {
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    void setProtos(SimProtos* hostProtos) {
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(protos), hostProtos, sizeof(SimProtos)));
    }

    uint iDivUp(uint a, uint b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

    void computeGridSize(uint n, uint blockSize, uint& numBlocks, uint& numThreads) {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    void calcHash(uint* gridParticleHash, uint* gridParticleIndex, float* pos, uint numParticles) {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // calculate the hash value of every sphere
        calcHashD <<< numBlocks, numThreads >>> (gridParticleHash, gridParticleIndex, (float3*)pos, numParticles);
        getLastCudaError("calcHashD execution failed.");
    }

    void sortParticles(uint* gridParticleHash, uint* gridParticleIndex, uint numParticles) {
        thrust::sort_by_key(
            thrust::device_ptr<uint>(gridParticleHash),
            thrust::device_ptr<uint>(gridParticleHash + numParticles),
            thrust::device_ptr<uint>(gridParticleIndex)
        );
    }

    void findCellStart(uint* cellStart, uint* cellEnd, uint* gridParticleHash, uint numParticles, uint numCells) {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells * sizeof(uint)));
        findCellStartD <<< numBlocks, numThreads >>> (cellStart, cellEnd, gridParticleHash, numParticles);
        getLastCudaError("findCellStartD execution failed.");
    }

    void collide(float* pos, float* vel, float* accel, uint* type,
        uint* gridParticleIndex,
        uint* cellStart, uint* cellEnd,
        uint numParticles,
        float deltaT
    ) {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // compute collision
        collideD <<< numBlocks, numThreads >>> ((float3*)accel, (float3*)pos, (float3*)vel, type, gridParticleIndex, cellStart, cellEnd, numParticles);
        getLastCudaError("collideD execution failed.");

        // update
        updateD <<< numBlocks, numThreads >>> ((float3*)pos, (float3*)vel, (float3*)accel, type, gridParticleIndex, numParticles, deltaT);
        getLastCudaError("updateD execution failed.");
    }

}
